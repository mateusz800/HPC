#include <stdio.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

__device__ 
double calculateDistance(double* data, double* center, int size){
    // euclidean distance
    double distance = 0;
    for(int i=0;i<size;i++){
        distance += pow(data[i] - center[i], 2);
    }
    return sqrt(distance);
}

__global__ 
void kmean(int offset, double* data, double* centers, int rows, int cols, int k, int* result, int* class_count){
    int id = blockIdx.x*blockDim.x+threadIdx.x +offset;
    if(id >= rows){
        return;
    }
    double distance;
    double min_distance;
    int min_distance_cluster;
    for(int i=0;i<k;i++){
        if(i == 0 ){
            min_distance = calculateDistance(&data[id*cols], &centers[i*cols], cols );
            min_distance_cluster = i;
        } else {
            distance = calculateDistance(&data[id*cols], &centers[i*cols], cols );
            if(distance < min_distance){
                min_distance = distance;
                min_distance_cluster = i;
            }
        }
    }
    //printf("cluster: %d\n", min_distance_cluster);
    result[id] = min_distance_cluster;
    atomicAdd(&class_count[min_distance_cluster], 1) ;

}


void loadData(char* file_url, int col_start, int cols, int rows, double* data_ptr){
    FILE* stream = fopen(file_url, "r");
    char line[1024];
    char* clipboard;

    // skip first row
    fgets(line, 1024, stream);

    for(int i=0;i<rows;i++){
        fgets(line, 1024, stream);
        clipboard = strtok( line, "," );
        int j = 0;
        while( clipboard != NULL )
        {
            if(j >= col_start){
                data_ptr[i * cols + j-col_start] = atof(clipboard);
            }
            clipboard = strtok( NULL, "," );
            j++;
            if(j >= col_start + cols){
                break;
            }
        }
    }
}

void saveResultAsCsv(int * data, int rows){
    FILE * file = fopen("result.csv", "w+");
    fprintf(file,"Id,Cluster\n");
    for(int i=0;i<rows;i++){
        fprintf(file, "%d, %d\n", i, data[i]);
    }
    fclose(file);
}
void saveClusterCentersAsCsv(double * data, int k, int cols){
    FILE * file = fopen("centers.csv", "w+");
    //fprintf(file,"Cluster id, Cluster\n");
    for(int i=0;i<k;i++){
        for(int j=0;j<cols;j++){
            if(j==cols-1){
                fprintf(file, "%f\n", data[i*cols+j]);
            } else {
                fprintf(file, "%f,", data[i*cols+j]);
            }
        }
    }
    fclose(file);
}


int main(int argc, char** argv){
    int k = 12;
     // weather - cols:6
    // iris - cols:2, rows:150, k: 3
    int rows = 1586822;
    int cols = 6;
    int steps = 30;
    int offset = 0;
    char* file_url = "wind_data_prepared.csv";

    size_t data_size = cols * rows * sizeof(double);
    size_t centers_size = k * cols * sizeof(double);
    size_t calc_classes_size = rows * sizeof(int);
    size_t class_count_size = k * sizeof(int);

    double * h_data = (double*) malloc(data_size); // data[row * cols + col]
    double * h_centers = (double *) malloc(centers_size); // center[k*cols + col]
    int * h_calc_classes = (int*) malloc(calc_classes_size);
    int * h_class_count = (int*) malloc(class_count_size);

   
    // load arguments
    if(argc == 5){
        file_url = argv[1];
        rows = atoi(argv[2]);
        cols = atoi(argv[3]);
        k = atoi(argv[4]);
    } else {
        printf("Continue with default parameters \n");
    }
        

    srand(time(NULL)); 
    loadData(file_url, 1, cols, rows, h_data);
    // initialize clusters centers
    for(int i=0; i < k ; i++){
        int data_index = (rand() % rows) *cols;
        for(int j=0;j<cols;j++){
            h_centers[i*cols +j] = h_data[data_index + j];
        }
       
    }

    double * d_data;
    double * d_centers;
    int * d_calc_classes;
    int * d_class_count;
    hipError_t error;
    dim3 blocksPerGrid(1024, 1, 1);
	dim3 threadsPerBlock(1024, 1, 1);
    int max_rows = blocksPerGrid.x * threadsPerBlock.x;
    hipMalloc(&d_data, data_size);
    hipMalloc(&d_centers, centers_size);
    hipMalloc(&d_calc_classes, calc_classes_size);
    hipMalloc(&d_class_count, class_count_size);
    hipMemcpy(d_data, h_data, data_size, hipMemcpyHostToDevice);

    for(int step=0;step<steps;step++){
        hipMemcpy(d_centers, h_centers, centers_size, hipMemcpyHostToDevice);
        hipMemset(d_class_count, 0, class_count_size);

        for(offset=0;offset<rows;offset+=max_rows){
            // call device function
            kmean<<<blocksPerGrid, threadsPerBlock>>>(offset, d_data, d_centers, rows, cols, k,  d_calc_classes, d_class_count);

            if(offset+max_rows < rows){
                calc_classes_size = max_rows * sizeof(int);
            } else{
                calc_classes_size = (rows - offset) * sizeof(int);
            }
         
            // send results to host 
            hipMemcpy(h_calc_classes + offset , d_calc_classes+offset, calc_classes_size, hipMemcpyDeviceToHost);
            hipMemcpy(h_class_count, d_class_count, class_count_size, hipMemcpyDeviceToHost);
            
            // check if error occured
            error = hipGetLastError();
            if(error != hipSuccess){
                fprintf(stderr, "Error: %s\n", hipGetErrorString(error));
                exit(-1);
            }
        }
       
        // recalculate clusters (mean value)
        if(step != steps-1){
            // reset cluster centers
            for(int i=0;i<k;i++){
                for(int j=0;j<cols;j++){
                    h_centers[i*cols+j] = 0;
                }
            }
            for(int i=0;i<rows;i++){
                // sum all values
                for(int j=0;j<cols;j++){
                    h_centers[h_calc_classes[i] * cols + j] += h_data[i*cols + j];
                }
     
               
            }
            for(int i=0;i<k;i++){
                for(int j=0;j<cols;j++){
                    if(h_class_count[i] != 0){
                        h_centers[i*cols + j] /= h_class_count[i];
                    }
                }
            }
        }
    }

    saveResultAsCsv(h_calc_classes, rows);
    saveClusterCentersAsCsv(h_centers, k, cols);

    hipFree(d_data);
    free(h_data);

    return 0;
}