#include <stdio.h>
#include <hip/hip_runtime.h>

/*
Application adds two vectors declared in the code
*/

__global__ void vecAdd(int* a, int* b , int* c, int size){
    // calculate thread id
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id < size){
        c[id] = a[id] + b[id];
    }
}

void printVector(int* vec, int size){
    printf("[%d", vec[0]);
    for(int i=1;i<size; i++){
        printf(", %d", vec[i]);
    }
    printf("]\n");
}

int main(int argc, char**argv){
    int size = 5;
    size_t vectorSize = size * sizeof(int);

    //initialize host variables
    int* h_vecA = (int*) malloc(vectorSize);
    int* h_vecB = (int*) malloc(vectorSize);
    int* h_vecResult = (int*) malloc(vectorSize);

    for(int i = 0; i < size; i++){
		h_vecA[i] = i;
		h_vecB[i] = i*i;
	}

    // initialize device variables
    int * d_vecA, *d_vecB, *d_vecResult;
    hipMalloc(&d_vecA, vectorSize);
    hipMalloc(&d_vecB, vectorSize);
    hipMalloc(&d_vecResult, vectorSize);

    hipMemcpy(d_vecA, h_vecA, vectorSize, hipMemcpyHostToDevice);
    hipMemcpy(d_vecB, h_vecB, vectorSize, hipMemcpyHostToDevice);

    dim3 blocksPerGrid(1, 1, 1);
	dim3 threadsPerBlock(size, 1, 1);

    vecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_vecA, d_vecB, d_vecResult, size);


    // copy the result to the device
    hipMemcpy(h_vecResult, d_vecResult, vectorSize, hipMemcpyDeviceToHost);
     

    printf("The result: \n");
    printVector(h_vecResult, size);

    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess)
    {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
        exit(-1);    
    }
    return 0;
}