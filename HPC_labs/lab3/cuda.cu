#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>



__host__
void printtime(struct timeval *start,struct timeval *stop) {
  long time=1000000*(stop->tv_sec-start->tv_sec)+stop->tv_usec-start->tv_usec;

  printf("\nCUDA execution time=%ld microseconds\n",time);

}

int main(int argc,char **argv) {

    struct timeval start,stop;

    gettimeofday(&start,NULL);

    // run your CUDA kernel(s) here



    // synchronize/finalize your CUDA computations

    gettimeofday(&stop,NULL);

    printtime(&start,&stop);


}
